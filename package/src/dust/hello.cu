#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>

#include <R.h>
#include <Rinternals.h>
#include <Rembedded.h>
#include <Rdefines.h>


__global__ void tau2_calc(int P, double *a, double *tau2, double lambda2, double sigma2)
{
    double _tau2=0;
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if( j<P)
    {
         hiprandState s ;
         // seed a random number generator
         hiprand_init ( (unsigned int)j , 0, 0, &s) ;
	            
         if (a[j] != 0.0 )
         {
              double InvTau2_1 = sqrt( lambda2 * sigma2)/fabs(a[j]);
             // double _tau2 = 1/func_invGau(InvTau2_1, lambda2);
	    
	          double y1 = hiprand_normal_double (&s);   
              double y = y1*y1;
              double x = InvTau2_1 + 0.5*InvTau2_1/lambda2 * ( InvTau2_1*y - sqrt(4*InvTau2_1*lambda2*y + InvTau2_1*InvTau2_1*y*y) );
	          if (  hiprand_uniform_double (&s) <= (InvTau2_1/(InvTau2_1+x)))
                  _tau2 = 1/x;
	          else
                  _tau2 = 1/((InvTau2_1*InvTau2_1)/x);
            
              tau2[j] = _tau2;
        }
        else
        {
            tau2[j] = 0.0;
        }        
    }  
}

#define THREADS_PER_BLOCK 64

void calculate_tau(double *a, double *tau2, double lambda2, double sigma2, int P)
{
    double *d_a, *d_tau2;

    hipMalloc( (void **) &d_a, P*sizeof(double) );
    hipMalloc( (void **) &d_tau2, P*sizeof(double) );
    hipMemcpy( d_a, a, P*sizeof(double), hipMemcpyHostToDevice );
    hipMemcpy( d_tau2, tau2, P*sizeof(double), hipMemcpyHostToDevice );

    tau2_calc<<< (P + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( P, d_a, d_tau2, lambda2, sigma2);
    
/*{
#hipError_t cudaerr = hipDeviceSynchronize();
#if ( cudaerr!=hipSuccess)
#Rprintf("kernel launch failed with err \"%s\".\n",hipGetErrorString(cudaerr));
#else
#Rprintf("Kernel launch good\n");
#}
*/

    hipMemcpy( tau2, d_tau2, P*sizeof(double), hipMemcpyDeviceToHost );

    hipFree( d_a );
    hipFree( d_tau2 );
}
