#include "hip/hip_runtime.h"
#include <R.h>
#include <Rinternals.h>
#include <Rembedded.h>
#include <Rdefines.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>

#include "gls_gpu.h"
#include "gpu_share.h" 
#include "gpu_matrix.cux"
#include "gpu_reduction.cux"

int _CheckCuda()
{
	int nCount = 0;
	if ( hipGetDeviceCount ( &nCount ) != hipSuccess )
    {
		Rprintf("Failed to call CUDA library, the GLS model may not run on GPU nodes.\n");
		return(0);
    }

    Rprintf("%d GPU card(s) is/are available for the GLS model.\n", nCount);
	return(nCount);
}

__device__ double _cuda_invGau(int seed, double theta, double chi)
{
    hiprandState s ;

    // seed a random number generator
    hiprand_init ( (unsigned int) seed, 0, 0, &s) ;
    double _rn = hiprand_normal_double (&s);   
    double _ru = hiprand_uniform_double (&s);

#ifdef USECUDA
#ifdef MONTIME
    _rn = 0.67;
    _ru = 0.45;
#endif
#endif

    //squared normal, i.e., chi-square with df=1
    double chisq1 = _rn * _rn;
    double y1    = theta + 0.5*theta/chi * ( theta*chisq1 - sqrt(4*theta*chi*chisq1 + theta*theta*chisq1*chisq1) );
    double y2    = theta*theta/y1;
    double out_1 = _ru < (theta/(theta+y1));
    double value = out_1*y1+(1-out_1)*y2;

    return(value);
}

 
__global__ void g_part0(int N, double rho, double tmp5 )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N)
    {
        printf("hello %d\n", i);   
    } 
}

int _cuda_gpart0( int N, double rho, double tmp5)
{   
    g_part0<<< 1, 5>>>(N, rho, tmp5 );
    hipDeviceSynchronize();
    ERRCHECK;
    
    return(0);
}
 
__device__ GPUShare* ConvetSharePoint( double* smb, int Q )
{
    GPUShare* p = (GPUShare*)smb; 
    p->tempMat0 = (double*)(&(p->pNext));
    p->tempMat1 = p->tempMat0 + (Q*Q+3);
    p->tempMat2 = p->tempMat1 + (Q*Q+3);
    p->tempMat3 = p->tempMat2 + (Q*Q+3);
    p->tempMat4 = p->tempMat3 + (Q*Q+3);
    p->tMA  = p->tempMat4 + (Q*Q+3);
    p->tMB  = p->tMA + (Q*Q+3);
    p->tMC  = p->tMB + (Q*Q+3);
    p->tMD  = p->tMC + (Q*Q+3);
    p->tME  = p->tMD + (Q*Q+3);

    p->tempMat0[0]= Q*Q;
    p->tempMat1[0]= Q*Q;
    p->tempMat2[0]= Q*Q;
    p->tempMat3[0]= Q*Q;
    p->tempMat4[0]= Q*Q;
    p->tMA[0]= Q*Q;
    p->tMB[0]= Q*Q;
    p->tMC[0]= Q*Q;
    p->tMD[0]= Q*Q;
    p->tME[0]= Q*Q;

    return(p);
}
        
void _initGPUobj(struct GPUobj* gCpu, struct GPUobj* gGpuObj, struct GPUobj* gGpuMap, unsigned int nsize, unsigned int N, unsigned int P, unsigned int Q, unsigned int NC )
{
printf("%p, %d, %d, %d, %d \n", nsize, N, P, Q, NC );

    memset(gCpu, 0, nsize);
    gCpu->nSize = nsize;
    gCpu->pNext = &(gCpu->pNext);

    gCpu->X     = make_matrix_ongpu( N, NC+1 );
    gCpu->Z     = make_matrix_ongpu( N, Q );
    gCpu->Z0    = make_matrix_ongpu( N, Q );
    gCpu->mInZ  = make_matrix_ongpu( N, 1 );
    gCpu->mu    = make_matrix_ongpu( LG, 1 );
    gCpu->alpha = make_matrix_ongpu( NC+1, LG );
    gCpu->a     = make_matrix_ongpu( P, LG );
    gCpu->a_old = make_matrix_ongpu( P, LG );
    gCpu->d     = make_matrix_ongpu( P, LG );
    gCpu->d_old = make_matrix_ongpu( P, LG );
    gCpu->vctP  = make_matrix_ongpu( P, 1 );
    gCpu->tau2  = make_matrix_ongpu( P, 1 );
    gCpu->tau2_x= make_matrix_ongpu( P, 1 );
    gCpu->gen_a = make_matrix_ongpu( P, N );
    gCpu->gen_d = make_matrix_ongpu( P, N );
    gCpu->tVN0  = make_matrix_ongpu( N, 1 );
    gCpu->tVN1  = make_matrix_ongpu( N, 1 );
    gCpu->tVN2  = make_matrix_ongpu( N, 1 );

printf("gCpu->X=%p\n", gCpu->X );
printf("gCpu->gen_a=%p \n", gCpu->gen_a );
printf("gCpu->gen_d=%p \n", gCpu->gen_d );
printf("Test %p +1 %p +2=%p\n", &(gCpu->pNext), &(gCpu->pNext) + 1, &(gCpu->pNext) + N  );

    gCpu->all_corTimes = make_matrix_list( (double**)(&(gCpu->pNext)), N, Q, Q );
    gCpu->all_corMat = make_matrix_list( (double**)( gCpu->all_corTimes + N), N, Q, Q );
    gCpu->all_corMat_MH = make_matrix_list( (double**)( gCpu->all_corMat + N), N, Q, Q );
    gCpu->all_corMat_Inv = make_matrix_list( (double**)( gCpu->all_corMat_MH + N), N, Q, Q );
    gCpu->all_corMat_MH_Inv = make_matrix_list( (double**)( gCpu->all_corMat_Inv + N), N, Q, Q );

printf("Test p1=%p p2=%p  p3==%p\n", gCpu->all_corTimes, gCpu->all_corMat, gCpu->all_corMat_MH  );

    gCpu->all_yi  = make_vector_list( (double**)( gCpu->all_corMat_MH_Inv + N), N, Q );
    gCpu->all_rd  = make_vector_list( (double**)( gCpu->all_yi + N), N, Q );
    gCpu->all_ui  = make_matrix_list( (double**)( gCpu->all_rd + N), N, Q, LG  );

    gCpu->tempMat  = make_matrix_list( (double**)(gCpu->all_ui + N), N, Q, Q  );
    gCpu->tmp2     = make_matrix_list( (double**)(gCpu->tempMat + N), N, LG, LG );
    gCpu->tmp3     = make_matrix_list( (double**)(gCpu->tmp2 + N), N, 1, LG );
    
    memcpy(gGpuMap, gCpu, nsize); 

printf("gCpu            =%p\n", gCpu);
printf("gGpuObj         =%p\n", gGpuObj);

    #define MAP_ADDR(x)  (double**)((char*)gGpuObj + (unsigned int)((char*)(x) - (char*)gCpu) )

    gGpuMap->all_corTimes = MAP_ADDR( gCpu->all_corTimes );
    gGpuMap->all_corMat = MAP_ADDR( gCpu->all_corMat );
    gGpuMap->all_corMat_MH = MAP_ADDR( gCpu->all_corMat_MH );
    gGpuMap->all_corMat_Inv = MAP_ADDR( gCpu->all_corMat_Inv );
    gGpuMap->all_corMat_MH_Inv = MAP_ADDR( gCpu->all_corMat_MH_Inv );
    gGpuMap->all_yi   = MAP_ADDR( gCpu->all_yi );
    gGpuMap->all_rd   = MAP_ADDR( gCpu->all_rd );
    gGpuMap->all_ui   = MAP_ADDR( gCpu->all_ui );

    gGpuMap->tempMat  = MAP_ADDR( gCpu->tempMat );
    gGpuMap->tmp2     = MAP_ADDR( gCpu->tmp2 );
    gGpuMap->tmp3     = MAP_ADDR( gCpu->tmp3 );

}

int Init_GPUobj(struct GPUobj** pCpuObj, struct GPUobj** pGpuObj, struct GPUobj** pGpuMap, int N, int P, int Q, int NC)
{
    int ngCudaSize = sizeof(struct GPUobj) + 50* N * sizeof(double*);

    //hipblasHandle_t h;
    //hipblasCreate(&h);
    //hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);
 
    struct GPUobj* gGpuObj;
    PERR( hipMalloc( (void **)&gGpuObj, ngCudaSize ) );
    *pGpuObj = gGpuObj;

    struct GPUobj* gCpuCopy = (struct GPUobj*)Calloc( ngCudaSize, char);
    *pCpuObj = gCpuCopy;

    struct GPUobj* gGpuMap = (struct GPUobj*)Calloc( ngCudaSize, char);
    *pGpuMap = gGpuMap;

    _initGPUobj( gCpuCopy, gGpuObj, gGpuMap, ngCudaSize, N, P, Q, NC );

    PERR( hipMemcpy( gGpuObj, gGpuMap, ngCudaSize, hipMemcpyHostToDevice ) );
    
printf("CPU copy=%p\n", *pCpuObj);
printf("GPU addr=%p\n", *pGpuObj);
printf("GPU map =%p\n", *gGpuMap);
    
    return(0);
}

void _freeGPUobj(struct GPUobj* pCpuObj, int N )
{
    PERR( hipFree( pCpuObj->X ) );
    PERR( hipFree( pCpuObj->Z ) );
    PERR( hipFree( pCpuObj->Z0 ) );
    PERR( hipFree( pCpuObj->mInZ ) );
    PERR( hipFree( pCpuObj->mu ) );
    PERR( hipFree( pCpuObj->alpha ) );
    PERR( hipFree( pCpuObj->a ) );
    PERR( hipFree( pCpuObj->d ) );
    PERR( hipFree( pCpuObj->a_old ) );
    PERR( hipFree( pCpuObj->d_old ) );
    PERR( hipFree( pCpuObj->vctP ) );
    PERR( hipFree( pCpuObj->tau2 ) );
    PERR( hipFree( pCpuObj->tau2_x ) );
    PERR( hipFree( pCpuObj->gen_a ) );
    PERR( hipFree( pCpuObj->gen_d ) );
    PERR( hipFree( pCpuObj->tVN0 ) );
    PERR( hipFree( pCpuObj->tVN1 ) );
    PERR( hipFree( pCpuObj->tVN2 ) );

    for(int i=0;i<N;i++)
    {
        PERR( hipFree( pCpuObj->all_corTimes[i] ) );
        PERR( hipFree( pCpuObj->all_corMat[i] ) );
        PERR( hipFree( pCpuObj->all_corMat_MH[i] ) );
        PERR( hipFree( pCpuObj->all_corMat_Inv[i] ) );
        PERR( hipFree( pCpuObj->all_corMat_MH_Inv[i] ) );
        PERR( hipFree( pCpuObj->all_yi[i] ) );
        PERR( hipFree( pCpuObj->all_rd[i] ) );
        PERR( hipFree( pCpuObj->all_ui[i] ) );
        PERR( hipFree( pCpuObj->tmp2[i] ) );
        PERR( hipFree( pCpuObj->tmp3[i] ) );
        PERR( hipFree( pCpuObj->tempMat[i] ) );
    }
    
}
    
int Free_GPUobj(struct GPUobj* pGpuObj, struct GPUobj* pCpuObj, struct GPUobj* gGpuMap, int N)
{
    _freeGPUobj( pCpuObj, N );

    PERR( hipFree(pGpuObj) );

    Free(gGpuMap); 

    Free(pCpuObj); 

    return(0);
}

__global__ void g_part1(struct GPUobj* gCuda, int N, double rho, double tmp5 )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < N)
    {
        int m = (int)_V( gCuda->mInZ, i) ;

        double* Z0 = gCuda->all_corTimes[i];

        for (int ii=0; ii<m; ii++)
          for (int jj=00; jj<m; jj++)
          {
              _M( gCuda->all_corMat[i], ii, jj ) =  pow(rho, _M(Z0, ii, jj));  
              _M( gCuda->all_corMat_MH[i], ii, jj ) =  pow(tmp5, _M(Z0, ii, jj));
          }    
    } 
}

int _cuda_gpart1( struct GPUobj* gCuda, struct GPUobj* gCpuObj, int N, double rho, double tmp5)
{   
//printf("part1 %p, %d, %f, %f %d, %d \n", gCuda, N, rho, tmp5, (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK);

    g_part1<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(gCuda, N, rho, tmp5 );
    hipDeviceSynchronize();
    ERRCHECK;

//printf("End of part1\n");
    // dont need copy all_corMat and all_corMat_MH back to CPU
    return(0);
}

__global__ void g_part2(struct GPUobj* gCuda, int N, int Q, int nC, double sigma2 )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;

    if(idx < N)
    {
#define tmp          gShare->tMA
#define tmp4         gShare->tMB
#define tui          gShare->tMC
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

        Matrix_Transpose_Col( col, gCuda->all_ui[idx], tui);
        __syncthreads();

        Matrix_mult_Double_Col( col, gCuda->all_corMat_Inv[idx], 1/sigma2, tempMat0 );
        __syncthreads();

        Matrix_mult_Matrix_Col( col, tempMat0, gCuda->all_ui[idx], tmp );
         __syncthreads();

        Matrix_Resize_Col( col, tmp4, 1.0,  Vector_GetLength(gCuda->all_rd[idx]), TRUE );
        __syncthreads();

       if(nC > 0)
            for( int nX = 0; nX < 1; nX++ )
            {
               Matrix_GetRow_Col( col, gCuda->alpha, nX, tempMat0 ); 
               __syncthreads();

               Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1);
               __syncthreads();

               Matrix_mult_Double_Col( col, tempMat1, _M(gCuda->X, idx, nX+1), tempMat1 );
               __syncthreads();

               Matrix_add_Matrix_Col( col, tempMat1, tmp4, tmp4 );
               __syncthreads();
            }
            
        __syncthreads();

        Matrix_Transpose_Col( col, gCuda->all_rd[idx], tempMat1 ); 
        __syncthreads();

        Matrix_sub_Matrix_Col( col, tempMat1, tmp4, tempMat1 );
        __syncthreads();

        //tmp3
        Matrix_mult_Matrix_Col( col, tempMat1, tmp, tempMat0);
        __syncthreads();
        Matrix_Copy_Col( col, gCuda->tmp3[idx], tempMat0);
        __syncthreads();

        //tmp2
        Matrix_mult_Matrix_Col( col, tui, tmp, tempMat0);
        __syncthreads();
        Matrix_Copy_Col( col, gCuda->tmp2[idx], tempMat0);
        __syncthreads();

#undef tempMat0 
#undef tempMat1 
#undef tempMat2 
#undef tmp
#undef tui
#undef tmp4
   }
}

int _cuda_gpart2( struct GPUobj* gCuda, struct GPUobj* gCpuObj, struct GPUobj* gGpuMap, int N, int Q, int nC, double sigma2, 
                 CFmMatrix& alpha, CFmMatrix& tmp2, CFmMatrix& tmp3 )
{
//printf("part2 %p, %d, %f, %f %d, %d \n", gCuda, N, nC, sigma2 );
    _copy_fmMatrix_Device( gCpuObj->alpha, alpha );
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);
    
    g_part2<<< N,  16, nShareSize >>>(gCuda, N, Q, nC, sigma2 );
    hipDeviceSynchronize();
    ERRCHECK;

    // tmp2 
    g_reduce_matrix( gGpuMap->tmp2, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp2, gCpuObj->tempMat[0]);

    // tmp3  
    g_reduce_matrix( gGpuMap->tmp3, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp3, gCpuObj->tempMat[0]);
    
//printf("End of part2\n");

    return(0);
}
 
__global__ void g_part3(struct GPUobj* gCuda, int N, int Q, int P )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;

    if(idx < N)
    {
#define ui           gShare->tMA
#define mean_effect  gShare->tMB
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

         Matrix_Resize_Col( col, mean_effect, Vector_GetLength( gCuda->all_yi[idx]), 1, TRUE );
        __syncthreads();
         
        Matrix_Copy_Col( col, ui, gCuda->all_ui[idx]);
        __syncthreads();
         
         for(int jj=0; jj < P; jj++)
         {
             double a0 = _M(gCuda->gen_a, jj, idx);
             if(  a0 != 0)
             {
                 Matrix_GetRow_Col( col, gCuda->a, jj, tempMat0 );
                 __syncthreads();

                 Matrix_Transpose_Col( col, tempMat0, tempMat1 );
                 __syncthreads();

                 Matrix_mult_Matrix_Col( col, ui, tempMat1, tempMat0 );
                 __syncthreads();

                 Matrix_mult_Double_Col( col, tempMat0, a0, tempMat0 );
                 __syncthreads();

                 Matrix_add_Matrix_Col( col, mean_effect, tempMat0, mean_effect); 
                 __syncthreads();
             }
             
             double d0 = _M(gCuda->gen_d, jj, idx);
             if( d0 != 0)
             {
                 Matrix_GetRow_Col( col, gCuda->d, jj, tempMat0 );
                 __syncthreads();

                 Matrix_Transpose_Col( col, tempMat0, tempMat1 );
                 __syncthreads();

                 Matrix_mult_Matrix_Col( col, ui, tempMat1, tempMat0 );
                 __syncthreads();

                 Matrix_mult_Double_Col( col, tempMat0, d0, tempMat0 );
                 __syncthreads();

                 Matrix_add_Matrix_Col( col, mean_effect, tempMat0, mean_effect ); 
                 __syncthreads();
             }   
         }
         
         Matrix_sub_Matrix_Col( col, gCuda->all_yi[idx], mean_effect, gCuda->all_rd[idx]);
         __syncthreads(); 

#undef ui         
#undef mean_effect
#undef tempMat0
#undef tempMat1
#undef tempMat2
    }
}
         
int _cuda_gpart3( struct GPUobj* gCuda, struct GPUobj* gCpuObj, int N, int Q, int P, CFmMatrix& a, CFmMatrix& d)
{
//printf("part3 %p, %d, %d\n", gCuda, N, P );
    _copy_fmMatrix_Device( gCpuObj->a, a );
    _copy_fmMatrix_Device( gCpuObj->d, d );
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);    
    g_part3<<< N, 16, nShareSize >>>( gCuda, N, Q, P );
    hipDeviceSynchronize();
    ERRCHECK;
    
//printf("End of part3\n");
    return (0);
}


__global__ void g_part4(struct GPUobj* gCuda, int N, int Q, int j, int nC, double sigma2 )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define tmp          gShare->tMA
#define tui          gShare->tMC
#define tmp4         gShare->tMB
#define ui           gShare->tMD
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

        Matrix_Resize_Col( col, tmp4, 1, Vector_GetLength(gCuda->all_rd[idx]), TRUE );
        __syncthreads();

        Matrix_Transpose_Col( col, gCuda->all_ui[idx], tui );
        __syncthreads();
        Matrix_Transpose_Col( col, tui, ui );
        __syncthreads();

        double a0 = _M(gCuda->gen_a, j, idx);
        if( a0 != 0.0)
        {
            if(nC > 0)
            {
                for(int nX = 0; nX < nC; nX++)
                {
                    Matrix_GetRow_Col( col, gCuda->alpha, nX, tempMat0 ); 
                    __syncthreads();
                    Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
                    __syncthreads();
                    Matrix_mult_Double_Col( col, tempMat1, _M(gCuda->X, idx, nX+1), tempMat1 );
                    __syncthreads();
                    Matrix_add_Matrix_Col( col, tempMat1, tmp4, tmp4 );
                    __syncthreads();
                }
             }
             
             Matrix_mult_Double_Col( col, gCuda->all_corMat_Inv[idx], a0/sigma2 , tempMat0);
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat0, ui, tmp);
             __syncthreads();
                 
             Matrix_Transpose_Col( col, gCuda->mu, tempMat0);
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
             __syncthreads();

             
             Matrix_Transpose_Col( col, gCuda->all_rd[idx], tempMat2); 
             __syncthreads();
             Matrix_sub_Matrix_Col( col, tempMat2, tempMat1, tempMat1);
             __syncthreads();
             Matrix_sub_Matrix_Col( col, tempMat1, tmp4, tempMat0);
             __syncthreads();

             Matrix_GetRow_Col( col, gCuda->a, j, tempMat1 );
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat1, tui, tempMat2);
             __syncthreads();
             Matrix_mult_Double_Col(col, tempMat2, a0, tempMat1);
             __syncthreads();

             Matrix_add_Matrix_Col( col, tempMat0, tempMat1, tempMat0 );
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat0, tmp , gCuda->tmp3[idx]);
             __syncthreads();
             
             Matrix_mult_Matrix_Col( col, tui, tmp, tempMat0);
             __syncthreads();
             Matrix_mult_Double_Col( col, tempMat0, a0, gCuda->tmp2[idx]); 
             __syncthreads();
        }
        else
        {
             Matrix_Resize_Col( col, gCuda->tmp2[idx], LG, LG, TRUE);
             __syncthreads();
             Matrix_Resize_Col( col, gCuda->tmp3[idx], 1, LG, TRUE);
             __syncthreads();
        }   

#undef tempMat0 
#undef tempMat1 
#undef tempMat2 
#undef tmp4
#undef tui         
#undef ui 
#undef tmp 
        
    }
}


int _cuda_gpart4( struct GPUobj* gCuda, struct GPUobj* gCpuObj, struct GPUobj* gGpuMap, int N, int Q, int j, int nC, double sigma2, 
                 CFmVector& mu, CFmMatrix& alpha, CFmMatrix& a, CFmMatrix& tmp2, CFmMatrix& tmp3)
{
//printf("part4 %p, %d, %d, %d \n", gCuda, N, j, nC );
    if (j==0)
    {
        _copy_fmMatrix_Device( gCpuObj->a, a);
        _copy_fmVector_Device( gCpuObj->mu, mu);
        _copy_fmMatrix_Device( gCpuObj->alpha, alpha);
    }
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);    
    g_part4<<< N, 16, nShareSize >>>(gCuda, N, Q, j, nC,sigma2 );
    hipDeviceSynchronize();
    ERRCHECK;

    // tmp2 and tmp 3 
    g_reduce_matrix( gGpuMap->tmp2, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp2, gCpuObj->tempMat[0] );

    g_reduce_matrix( gGpuMap->tmp3, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp3, gCpuObj->tempMat[0] );

//printf("End of part4\n");

    return(0);
}


__global__ void g_part5(struct GPUobj* gCuda, int N, int Q, int j )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define ui       gShare->tMA
#define rd       gShare->tMB
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

        Matrix_Copy_Col( col, ui, gCuda->all_ui[idx] );
        Matrix_Copy_Col( col, rd, gCuda->all_rd[idx] );
        __syncthreads();

        double gen_a = _M(gCuda->gen_a, j, idx);  
        if(  gen_a != 0.0 )
        {
             Matrix_GetRow_Col( col, gCuda->a_old, j, tempMat0);
             __syncthreads();
             Matrix_Transpose_Col( col, tempMat0, tempMat1);
             __syncthreads();
             Matrix_mult_Matrix_Col( col, ui, tempMat1, tempMat2);
             __syncthreads();
             
             //to save up transfer time, here only transfer one vector into VN0
             //Matrix_GetRow_Col( col, gCuda->a, j, tempMat0);
             //__syncthreads();
             //Matrix_Transpose_Col( col, tempMat0, tempMat1 );
             //__syncthreads();
             
             Matrix_Copy_Col( col, tempMat1, gCuda->tVN0);
             __syncthreads();
             Matrix_mult_Matrix_Col( col, ui, tempMat1, tempMat0);
             __syncthreads();
             Matrix_sub_Matrix_Col( col, tempMat2, tempMat0, tempMat0);
             __syncthreads();
             
             Matrix_mult_Double_Col( col, tempMat0, gen_a, tempMat0);
             __syncthreads();
         
             Matrix_add_Matrix_Col( col, rd, tempMat0, gCuda->all_rd[idx] );
             __syncthreads();
        }

#undef ui       
#undef rd       
#undef tempMat0     
#undef tempMat1     
#undef tempMat2     

    }
}

int _cuda_gpart5( struct GPUobj* gCuda, struct GPUobj* gCpuObj, int N, int Q, int j, CFmMatrix& a, CFmMatrix& a_old)
{
    //only fully update a and a_old at the first time to save up computational times.
    if(j==0)
        _copy_fmMatrix_Device( gCpuObj->a_old, a_old);

    CFmVector new_aj(LG, 0); 
    new_aj = a.GetRow(j);
    _copy_fmVector_Device(gCpuObj->tVN0, new_aj );
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);

    g_part5<<< N, 16, nShareSize >>>(gCuda, N, Q, j );
    hipDeviceSynchronize();
    ERRCHECK;

//printf("End of part5\n");

    return(0);
} 



__global__ void g_part6(struct GPUobj* gCuda, int P, double sigma2, double lambda2, double lambda2_x)
{    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < P )
    {
#define tau2    gCuda->tau2
#define tau2_x  gCuda->tau2_x

        double vp = _V( gCuda->vctP, i ); 
        if( vp == 2.0)
        {
            double InvTau2_1 = sqrt( LG * lambda2 * sigma2/ Matrix_RowProd( gCuda->a, i, i));
            _V( tau2, i) = 1/_cuda_invGau( i*10, InvTau2_1, LG * lambda2);
            _V( tau2_x, i) = 0.0;
         }
         else if(vp == 1.0)
         {
             double InvTau2_1 = sqrt( LG * lambda2_x * sigma2/Matrix_RowProd( gCuda->a, i, i) );
             _V( tau2_x, i) = 1/_cuda_invGau( i*10+1, InvTau2_1, LG*lambda2_x);
             _V( tau2, i) = 0.0;
         }
         else
         {
             _V( tau2, i) = 0.0;
             _V( tau2_x, i) = 0.0;
         }

#undef tau2
#undef tau2_x
    }
}

int _cuda_gpart6( struct GPUobj* gCuda, struct GPUobj* gCpuObj, int P, double sigma2, double lambda2, double lambda2_x, 
                 CFmVector& vctP, CFmMatrix& a, CFmVector& tau2, CFmVector& tau2_x )
{
    _copy_fmVector_Device( gCpuObj->vctP, vctP);
    _copy_fmMatrix_Device( gCpuObj->a, a);
    
    g_part6<<< (P + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( gCuda, P, sigma2, lambda2, lambda2_x );
    hipDeviceSynchronize();
    ERRCHECK;
    
    // tau2 and tau2_x
    _copyback_fmVector_Host(tau2, gCpuObj->tau2 );
    _copyback_fmVector_Host(tau2_x, gCpuObj->tau2_x );
    
    return(0);
}

__global__ void g_part7(struct GPUobj* gCuda, int N, int Q, int j, int nC, double sigma2)
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define tmp          gShare->tMA
#define tmp4         gShare->tMB
#define tui          gShare->tMC
#define gtmp2        gShare->tMD 
#define gtmp3        gShare->tME
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

        Matrix_Resize_Col( col, tmp4, 1, Vector_GetLength(gCuda->all_rd[idx]), TRUE );
        Matrix_Resize_Col( col, gtmp2, LG, LG, TRUE );
        Matrix_Resize_Col( col, gtmp3, 1, LG, TRUE );
        Matrix_Transpose_Col( col, gCuda->all_ui[idx], tui );
        __syncthreads();

        double d0 = _M(gCuda->gen_d, j, idx);
        if( d0 != 0.0 )
        {
             if(nC > 0)
             {
                 for(int nX = 0; nX < nC; nX++)
                 {
                     Matrix_GetRow_Col( col, gCuda->alpha, nX, tempMat0); 
                     __syncthreads();
                     Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1);
                     __syncthreads();
                     Matrix_mult_Double_Col( col, tempMat1, _M( gCuda->X, idx, nX+1), tempMat1 );
                     __syncthreads();
                     Matrix_add_Matrix_Col( col, tempMat1, tmp4, tmp4);
                     __syncthreads();
                 }
             }
                 
             Matrix_mult_Double_Col( col, gCuda->all_corMat_Inv[idx], d0/sigma2, tempMat0 );
             __syncthreads();
             Matrix_Transpose_Col( col, tui, tempMat1 );
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat0, tempMat1, tmp);
             __syncthreads();
                 
             Matrix_Transpose_Col( col, gCuda->mu, tempMat0 );
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
             __syncthreads();
             Matrix_Transpose_Col( col, tempMat1, tempMat0 );
             __syncthreads();

             Matrix_Transpose_Col( col, tmp4, tempMat1 );
             __syncthreads();

             Matrix_sub_Matrix_Col( col, gCuda->all_rd[idx], tempMat1, tempMat1 );
             __syncthreads();
             Matrix_sub_Matrix_Col( col, tempMat1, tempMat0, tempMat0 );
             __syncthreads();

             Matrix_GetRow_Col( col, gCuda->d, j, tempMat2 );
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat2, tui, tempMat1 );
             __syncthreads();
             Matrix_mult_Double_Col( col, tempMat1, d0, tempMat1 );
             __syncthreads();
             Matrix_Transpose_Col( col, tempMat1, tempMat2 );
             __syncthreads();
    
             Matrix_add_Matrix_Col( col, tempMat0, tempMat2, tempMat1 );
             __syncthreads();
             Matrix_Transpose_Col( col, tempMat1, tempMat0 );
             __syncthreads();
             Matrix_mult_Matrix_Col( col, tempMat0, tmp, gtmp3);
             __syncthreads();
             
             Matrix_mult_Matrix_Col( col, tui, tmp, tempMat0 );
             __syncthreads();
             Matrix_mult_Double_Col( col, tempMat0, d0,  gtmp2);
             __syncthreads();
        }

        Matrix_Copy_Col( col, gCuda->tmp2[idx], gtmp2);
        __syncthreads();
        Matrix_Copy_Col( col, gCuda->tmp3[idx], gtmp3);
        __syncthreads();


#undef tmp          
#undef tmp4       
#undef tui      
#undef tempMat0     
#undef tempMat1
#undef tempMat2
#undef gtmp2
#undef gtmp3

    }
}


int _cuda_gpart7( struct GPUobj* gCuda, struct GPUobj* gCpuObj, struct GPUobj* gGpuMap, int N, int Q, int j, int nC, double sigma2, 
                 CFmMatrix& alpha, CFmVector& mu, CFmMatrix& d, CFmMatrix& tmp2, CFmMatrix& tmp3 )
{
    if (j==0)
    {
        _copy_fmMatrix_Device( gCpuObj->alpha, alpha);
        _copy_fmMatrix_Device( gCpuObj->d, d);
        _copy_fmVector_Device( gCpuObj->mu, mu);
    }
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);
    g_part7<<< N, 16, nShareSize>>>(gCuda, N, Q, j, nC, sigma2 );
    hipDeviceSynchronize();
    ERRCHECK;

    // tmp2 and tmp3 
    g_reduce_matrix( gGpuMap->tmp2, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp2, gCpuObj->tempMat[0] );

    g_reduce_matrix( gGpuMap->tmp3, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp3, gCpuObj->tempMat[0]);
    
    return(0);
}


__global__ void g_part8( struct GPUobj* gCuda, int N, int Q, int j )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define ui           gShare->tMA
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();
        
        Matrix_Copy_Col( col, ui, gCuda->all_ui[idx] );
        __syncthreads();

        double d0 = _M(gCuda->gen_d, j, idx );
        if( d0!= 0)
        {
             Matrix_GetRow_Col( col, gCuda->d_old, j, tempMat0 );
             __syncthreads();
             Matrix_Transpose_Col( col, tempMat0, tempMat1 );
            __syncthreads();
             Matrix_mult_Matrix_Col( col, ui, tempMat1, tempMat0);
            __syncthreads();

             // Save time of data transfer.
             //Matrix_GetRow_Col( col, gCuda->d, j, tempMat1 );
             //__syncthreads();
             //Matrix_Transpose_Col( col, tempMat1, tempMat2 );
             //__syncthreads();

             Matrix_Copy_Col( col, tempMat2, gCuda->tVN0);
             __syncthreads();
             Matrix_mult_Matrix_Col( col, ui, tempMat2, tempMat1 );
             __syncthreads();
             
             Matrix_sub_Matrix_Col( col, tempMat0, tempMat1, tempMat0 );
             __syncthreads();
             Matrix_Transpose_Col( col, tempMat0, tempMat1);
            __syncthreads();
             
             Matrix_mult_Double_Col( col, tempMat1, d0, tempMat1 );
             __syncthreads();

             Matrix_Transpose_Col( col, tempMat1, tempMat0 );
             __syncthreads();
         
             Matrix_add_Matrix_Col( col, gCuda->all_rd[idx], tempMat0, gCuda->all_rd[idx]);
             __syncthreads();
        }
#undef ui      
#undef tempMat0     
#undef tempMat1
#undef tempMat2
    }
}

int _cuda_gpart8( struct GPUobj* gCuda, struct GPUobj* gCpuObj, int N, int Q, int j, CFmMatrix& d, CFmMatrix& d_old)
{
    if(j==0)
        _copy_fmMatrix_Device( gCpuObj->d_old, d_old );
        

    //_copy_fmMatrix_Device( gCpuObj->d, d );
    CFmVector new_dj(LG, 0); 
    new_dj = d.GetRow(j);
    _copy_fmVector_Device(gCpuObj->tVN0, new_dj );
    
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);
    g_part8<<< N, 16, nShareSize >>>( gCuda, N, Q, j );
    hipDeviceSynchronize();
    ERRCHECK;

    return(0);
}


__global__ void g_part9(struct GPUobj* gCuda, int P, double lambda2_st, double lambda2_st_x, double sigma2 )
{    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if( i < P )
    {
#define tau2_st    gCuda->tau2
#define tau2_st_x  gCuda->tau2_x

        double vp = _V( gCuda->vctP, i ); 
        if( vp == 2.0)
        {
            double InvTau2_1 = sqrt( LG * lambda2_st * sigma2/ Matrix_RowProd(gCuda->d, i, i));
            _V( tau2_st, i) = 1/_cuda_invGau( i*10, InvTau2_1, LG * lambda2_st);
            _V( tau2_st_x, i) = 0.0;
        }
        else if( vp == 1.0)
        {
            double InvTau2_1 = sqrt( LG * lambda2_st_x * sigma2/Matrix_RowProd(gCuda->d, i, i) );
            _V( tau2_st_x, i) = 1/_cuda_invGau( i*10+1, InvTau2_1, LG*lambda2_st_x);
            _V( tau2_st, i) = 0;
        }
        else
        {
            _V( tau2_st, i) = 0;
            _V( tau2_st_x, i) = 0;
        }
#undef tau2_st
#undef tau2_st_x 
    }
}


int _cuda_gpart9( struct GPUobj* gCuda, struct GPUobj* gCpuObj, int P, double lambda_st2, double lambda_st2_x, double sigma2, 
                 CFmVector& vctP, CFmMatrix& d, CFmVector& tau2_st, CFmVector& tau2_st_x  )
{
    _copy_fmVector_Device( gCpuObj->vctP, vctP );
    _copy_fmMatrix_Device( gCpuObj->d, d );

    g_part9<<< (P + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( gCuda, P, lambda_st2, lambda_st2_x, sigma2 );
    hipDeviceSynchronize();
    ERRCHECK;
    
    _copyback_fmVector_Host( tau2_st, gCpuObj->tau2);
    _copyback_fmVector_Host( tau2_st_x, gCpuObj->tau2_x);
   
    return(0);
}


__global__ void g_part10( struct GPUobj* gCuda, int N, int Q, int nC, int nX, double sigma2 )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define tui      gShare->tMA
#define ui       gShare->tMB
#define tmp      gShare->tMC
#define tmp4     gShare->tMD 
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2

        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

         Matrix_Resize_Col( col, tmp4, 1.0, Vector_GetLength( gCuda->all_rd[idx] ), TRUE );
         Matrix_Copy_Col( col, ui, gCuda->all_ui[idx]);
        __syncthreads();

         Matrix_Transpose_Col( col, ui, tui );
        __syncthreads();

         if(nC > 0)
            for( int nX2 = 0; nX2 < nC; nX2++ )
            {
                if(nX2 != nX)
                {
                    Matrix_GetRow_Col( col, gCuda->alpha, nX2, tempMat0 ); 
                   __syncthreads();
                    
                    Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
                    __syncthreads();
                    
                    Matrix_mult_Double_Col( col, tempMat1, _M( gCuda->X, idx, nX2 + 1), tempMat1 );
                    __syncthreads();
                    
                    Matrix_add_Matrix_Col( col, tmp4, tempMat1, tmp4 );
                    __syncthreads();
                }  
            }
            
         double x0 = _M( gCuda->X, idx, nX+1);   
         
         Matrix_mult_Double_Col( col, gCuda->all_corMat_Inv[idx], 1/sigma2, tempMat0 );
         __syncthreads();
         Matrix_mult_Double_Col( col, tempMat0, x0, tempMat1 );
         __syncthreads();
         Matrix_mult_Matrix_Col( col, tempMat1, ui, tmp );
         __syncthreads();

         Matrix_Transpose_Col( col, gCuda->mu, tempMat0 );
         __syncthreads();
         Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
         __syncthreads();

         Matrix_Transpose_Col( col, gCuda->all_rd[idx], tempMat0 );
         __syncthreads();

         Matrix_sub_Matrix_Col( col, tempMat0, tempMat1, tempMat0 );
         __syncthreads();
         Matrix_sub_Matrix_Col( col, tempMat0, tmp4, tempMat0 );
         __syncthreads();
         Matrix_mult_Matrix_Col( col, tempMat0, tmp, gCuda->tmp3[idx] );
         __syncthreads();
  
         Matrix_mult_Matrix_Col( col, tui, tmp, tempMat0 );
         __syncthreads();
         Matrix_mult_Double_Col( col, tempMat0, x0, gCuda->tmp2[idx]);
         __syncthreads();

#undef tui         
#undef ui         
#undef rd       
#undef tmp  
#undef tmp4 
#undef tempMat0        
#undef tempMat1        
#undef tempMat2
     }
}

int _cuda_gpart10( struct GPUobj* gCuda, struct GPUobj* gCpuObj, GPUobj* gGpuMap, int N, int Q, int nC, int nX, double sigma2, 
                  CFmMatrix& alpha, CFmVector& mu, CFmMatrix& tmp2, CFmMatrix& tmp3 )
{
    _copy_fmMatrix_Device( gCpuObj->alpha, alpha);
    _copy_fmVector_Device( gCpuObj->mu, mu);
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);
    g_part10<<< N, 16, nShareSize >>>( gCuda, N, Q, nC, nX, sigma2 );
    hipDeviceSynchronize();
    ERRCHECK;

    // tmp2 
    g_reduce_matrix( gGpuMap->tmp2, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp2, gCpuObj->tempMat[0]);
    
    // tmp3
    g_reduce_matrix( gGpuMap->tmp3, gGpuMap->tempMat, N );
    _copyback_fmMatrix_Host( tmp3, gCpuObj->tempMat[0]);

    return(0);
}


__global__ void g_part11( struct GPUobj* gCuda, int N, int Q, int nC )
{    
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define sigma2_scale gCuda->tVN0
#define tmp4         gShare->tMA
#define tmpv_sigma2  gShare->tMB
#define tmpv_get2    gShare->tMC
#define tui          gShare->tMD 
#define tempMat0     gShare->tempMat0
#define tempMat1     gShare->tempMat1
#define tempMat2     gShare->tempMat2


        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

        Matrix_Resize_Col( col, tmp4, 1, Vector_GetLength(gCuda->all_rd[idx]), TRUE);
        Matrix_Transpose_Col( col, gCuda->all_ui[idx], tui );
        __syncthreads();
        
        if(nC > 0)
        {
            for(int nX = 0; nX < nC; nX++)
            {
                Matrix_GetRow_Col( col, gCuda->alpha, nX, tempMat0 );
                __syncthreads();
                Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
                __syncthreads();
                Matrix_mult_Double_Col( col, tempMat1, _M(gCuda->X, idx, nX+1), tempMat1 );
                __syncthreads();
                Matrix_add_Matrix_Col( col, tempMat1, tmp4, tmp4);
                __syncthreads();
            }
        }
        
        Matrix_Transpose_Col( col, gCuda->mu, tempMat0 );
        __syncthreads();
        Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1 );
        __syncthreads();
        
        Matrix_Transpose_Col( col,  gCuda->all_rd[idx], tempMat0 );
        __syncthreads();
        Matrix_sub_Matrix_Col( col,  tempMat0, tempMat1, tempMat0 );
        __syncthreads();
        Matrix_sub_Matrix_Col( col,  tempMat0, tmp4, tmpv_sigma2 );
        __syncthreads();
        
        Matrix_mult_Matrix_Col( col, tmpv_sigma2, gCuda->all_corMat_Inv[idx], tempMat0 );
        __syncthreads();
        Matrix_Transpose_Col( col,  tmpv_sigma2, tempMat1 );
        __syncthreads();
        Matrix_mult_Matrix_Col( col, tempMat0, tempMat1, tmpv_get2 );
        __syncthreads();
        
        _V(sigma2_scale, idx ) = _M( tmpv_get2, 0, 0);

#undef sigma2_scale 
#undef tmp4         
#undef tmpv_sigma2  
#undef tmpv_get2    
#undef tui          
#undef tempMat0     
#undef tempMat1     
#undef tempMat2     

    }
}

int _cuda_gpart11( struct GPUobj* gCuda, struct GPUobj* gCpuObj, struct GPUobj* gGpuMap, int N, int Q, int nC, 
                  CFmMatrix& alpha, CFmVector& mu, double* sigma2_scale)
{
    _copy_fmMatrix_Device( gCpuObj->alpha, alpha);
    _copy_fmVector_Device( gCpuObj->mu, mu);
    
    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);
    g_part11<<< N, 16, nShareSize >>>( gCuda, N, Q, nC );
    hipDeviceSynchronize();
    ERRCHECK;
    
    //gCuda->tVA <--> sigma2_scale
    g_reduce_double( gGpuMap->tVN0, gGpuMap->tVN1, N, sigma2_scale);

    if(1) 
    { 
        double dRet = g_reduce_double_test( gGpuMap->tVN0, N );
        if( (dRet - *sigma2_scale)> 1e-6 )
           printf("Part11 Failed to reduce(sigma2_scale C:%.5f, G:%.5f)\n", dRet, *sigma2_scale );
    }

    return(0);
}

__global__ void g_part12(struct GPUobj* gCuda, int N, int Q, int nC, double sigma2 )
{
    extern __shared__ double smb[];

    //int i = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.x;
    int idx = blockIdx.x;
    
    if(idx < N)
    {
#define exp_diff  gCuda->tVN0
#define tmp4      gShare->tMA
#define tui       gShare->tMB
#define exp_mat   gShare->tMC
#define tempMat0  gShare->tempMat0
#define tempMat1  gShare->tempMat1
#define tempMat2  gShare->tempMat2
#define allRd_i   gShare->tempMat4


        GPUShare* gShare = (GPUShare*)smb;
        if(col==0) gShare = ConvetSharePoint( smb, Q );
        __syncthreads();

        Matrix_Resize_Col( col, tmp4, 1, Vector_GetLength(gCuda->all_rd[idx]), TRUE);
        Matrix_Transpose_Col( col, gCuda->all_ui[idx], tui );
        __syncthreads();

        if(nC > 0)
        {
            for(int nX = 0; nX < nC; nX++)
            {
                Matrix_GetRow_Col( col, gCuda->alpha, nX, tempMat0);
                __syncthreads();
                Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat1);
                __syncthreads();
                Matrix_mult_Double_Col( col, tempMat1, _M( gCuda->X, idx, nX+1), tempMat1 );
                __syncthreads();
                Matrix_add_Matrix_Col( col, tmp4, tempMat1, tmp4 );
                __syncthreads();
            }
        }
        
        Matrix_Transpose_Col( col, gCuda->mu, tempMat0 );
        Matrix_mult_Matrix_Col( col, tempMat0, tui, tempMat2);

        Matrix_Transpose_Col( col, gCuda->all_rd[idx], tempMat0);
        Matrix_sub_Matrix_Col( col, tempMat0, tempMat2, tempMat0);
        Matrix_sub_Matrix_Col( col, tempMat0, tmp4, allRd_i );

        Matrix_mult_Double_Col( col, gCuda->all_corMat_MH_Inv[idx], 1/sigma2, tempMat0);
        Matrix_mult_Matrix_Col( col, allRd_i, tempMat0, tempMat1 );
        Matrix_Transpose_Col(col, allRd_i, tempMat2); 
        Matrix_mult_Matrix_Col( col, tempMat1, tempMat2, exp_mat );

        Matrix_mult_Double_Col( col, gCuda->all_corMat_Inv[idx], 1/sigma2, tempMat0);
        Matrix_mult_Matrix_Col( col, allRd_i, tempMat0, tempMat1 );
        Matrix_Transpose_Col( col, allRd_i, tempMat2); 
        Matrix_mult_Matrix_Col( col, tempMat1, tempMat2, tempMat0 );

        Matrix_sub_Matrix_Col( col, exp_mat, tempMat0, exp_mat );

        _V( exp_diff, idx ) = _M( exp_mat, 0, 0 );


#undef exp_diff  
#undef tmp4      
#undef tui       
#undef exp_mat   
#undef tempMat0  
#undef tempMat1  
#undef tempMat2  
#undef allRd_i   

    }
}

int _cuda_gpart12( struct GPUobj* gCuda, struct GPUobj* gCpuObj, struct GPUobj* gGpuMap, int N, int Q, int nC, double sigma2, 
                  CFmMatrix& alpha, CFmVector& mu, double* exp_diff)
{
    _copy_fmMatrix_Device( gCpuObj->alpha, alpha);
    _copy_fmVector_Device( gCpuObj->mu, mu);

    int nShareSize = ((Q*Q+3)*10 + (Q+3)*10 ) * sizeof(double);
    g_part12<<< N, 16, nShareSize >>>( gCuda, N, Q, nC, sigma2 );
    hipDeviceSynchronize();
    ERRCHECK;

    g_reduce_double( gGpuMap->tVN0, gGpuMap->tVN1, N, exp_diff);
    
    return(0);
}

__global__ void g_showTmp3(struct GPUobj* gCuda, int N )
{    
    for(int i=0; i<N; i++)
    {
        Print_Matrix( gCuda->tmp3[i] );
    }
}
//g_showTmp3<<< 1,  1>>> (gCuda, N);

