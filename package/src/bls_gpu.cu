#include "hip/hip_runtime.h"
#include <R.h>
#include <Rinternals.h>
#include <Rembedded.h>
#include <Rdefines.h>

#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdio.h>

#include "bls_gpu.h"

#include "gpu_share.h" 

__device__ double _cuda_invGau_same(int seed, double theta, double chi)
{
    hiprandState s ;

    // seed a random number generator
    hiprand_init ( (unsigned int) seed, 0, 0, &s) ;
    double _rn = hiprand_normal_double (&s);   
    double _ru = hiprand_uniform_double (&s);

#ifdef USECUDA
#ifdef MONTIME
    _rn = 0.67;
    _ru = 0.45;
#endif
#endif

    //squared normal, i.e., chi-square with df=1
    double chisq1 = _rn * _rn;
    double y1    = theta + 0.5*theta/chi * ( theta*chisq1 - sqrt(4*theta*chi*chisq1 + theta*theta*chisq1*chisq1) );
    double y2    = theta*theta/y1;
    double out_1 = _ru < (theta/(theta+y1));
    double value = out_1*y1+(1-out_1)*y2;

    return(value);
}


void _initBlsGPU(struct blsGPUobj* gCpu, struct blsGPUobj* gGpuObj, unsigned int nsize, unsigned int N, unsigned int P )
{
    memset(gCpu, 0, nsize);
    gCpu->nSize = nsize;
    gCpu->pNext = &(gCpu->pNext);

    gCpu->gen_pA  = make_matrix_ongpu( N, P );
    gCpu->gen_pD  = make_matrix_ongpu( N, P );
    gCpu->ad      = make_matrix_ongpu( P, 1 );
    gCpu->Var     = make_matrix_ongpu( P, 1 );
    gCpu->tmp     = make_matrix_ongpu( P, 1 );
    gCpu->tau2    = make_matrix_ongpu( P, 1 );
    gCpu->tmp3    = make_matrix_ongpu( N, 1 );
    gCpu->spdY    = make_matrix_ongpu( N, 1 );
}

int Init_blsGPU(struct blsGPUobj** pCpuObj, struct blsGPUobj** pGpuObj, int N, int P )
{
    int ngCudaSize = sizeof(struct blsGPUobj) ;

    struct blsGPUobj* gGpuObj;
    PERR( hipMalloc( (void **)&gGpuObj, ngCudaSize ) );
    *pGpuObj = gGpuObj;

    struct blsGPUobj* gCpuCopy = (struct blsGPUobj*)Calloc( ngCudaSize, char);
    *pCpuObj = gCpuCopy;

    _initBlsGPU( gCpuCopy, gGpuObj, ngCudaSize, N, P );

    PERR( hipMemcpy( gGpuObj, gCpuCopy, ngCudaSize, hipMemcpyHostToDevice ) );
    
printf("CPU copy=%p\n", *pCpuObj);
printf("GPU addr=%p\n", *pGpuObj);
    
    return(0);
}

void _freeBlsGPU(struct blsGPUobj* pCpuObj, int N )
{
    PERR( hipFree( pCpuObj->gen_pA ) );
    PERR( hipFree( pCpuObj->gen_pD ) );
    PERR( hipFree( pCpuObj->ad ) );
    PERR( hipFree( pCpuObj->Var ) );
    PERR( hipFree( pCpuObj->tau2 ) );
    PERR( hipFree( pCpuObj->tmp ) );
    PERR( hipFree( pCpuObj->spdY ) );
    PERR( hipFree( pCpuObj->tmp3 ) );
}
    
int Free_blsGPU(struct blsGPUobj* pGpuObj, struct blsGPUobj* pCpuObj, int N)
{
    _freeBlsGPU( pCpuObj, N );

    PERR( hipFree(pGpuObj) );

    Free(pCpuObj); 

    return(0);
}

__global__ void b_part1(struct blsGPUobj* gCuda, int N, int P )
{    
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j < P)
    {
        double a_j = _V(gCuda->ad, j );
        double aMu_j = 0.0;
        int N_AA = 0;
        int N_aa = 0;

        for (int k=0;k<N;k++)
        {
             double pA_kj = _M(gCuda->gen_pA, k, j);
             double spdY_k = _V(gCuda->spdY, k );
             
             double x0 = _V( gCuda->tmp3, k) - spdY_k + pA_kj * a_j;
             aMu_j = aMu_j + x0 * pA_kj;

             if (pA_kj>0) N_AA++;
             if (pA_kj<0) N_aa++;
        }     
        
        aMu_j = aMu_j / _V( gCuda->tmp, j);
        double aVar_j = _V(gCuda->Var, j);
        
        //repalce rnorm with hiprand_normal_double
        //double new_a = rnorm( aMu_j, sqrt(aVar_j) );
        
        hiprandState s ;
        // seed a random number generator
        hiprand_init ( (unsigned int) j*100, 0, 0, &s) ;
        double _rn = hiprand_normal_double (&s);   
        double new_a = _rn * sqrt(aVar_j) + aMu_j;
        
        if (N_AA==0 || N_aa==0) 
            new_a = 0.0;

        _V(gCuda->ad, j) = new_a;

   }
}


int _cuda_bpart1( struct blsGPUobj* gCuda, struct blsGPUobj* gCpuObj, int N, int P, CFmVector& spdY, CFmVector& aVar, CFmVector& tmp, CFmVector& tmp3, CFmVector& a)
{
    _copy_fmVector_Device( gCpuObj->spdY, spdY);
    _copy_fmVector_Device( gCpuObj->tmp, tmp );
    _copy_fmVector_Device( gCpuObj->tmp3, tmp3 );
    _copy_fmVector_Device( gCpuObj->Var, aVar);
    _copy_fmVector_Device( gCpuObj->ad, a );
    
    b_part1<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(gCuda, N, P);
    hipDeviceSynchronize();
    ERRCHECK;

    _copyback_fmVector_Host( a, gCpuObj->ad);
    
//printf("End of part1\n");
    return (0);
}

__global__ void b_part2(struct blsGPUobj* gCuda, int P, double lambda2, double sigma2 )
{    
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j < P)
    {
        double _tau2 = 0.0;
        double aj = _V(gCuda->ad, j);
        if ( aj != 0.0 )
        {
            double InvTau2_1 = sqrt( lambda2 * sigma2)/fabs(aj);
            _tau2 = 1/ _cuda_invGau_same(j*15,  InvTau2_1, lambda2);
        }   
        _V(gCuda->tau2, j) = _tau2;
   }
}

int _cuda_bpart2( struct blsGPUobj* gCuda, struct blsGPUobj* gCpuObj, int P, double lambda2, double sigma2, CFmVector& a, CFmVector& tau2 )
{
    _copy_fmVector_Device( gCpuObj->ad, a );
    
    b_part2<<< (P + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( gCuda, P, lambda2, sigma2  );
    hipDeviceSynchronize();
    ERRCHECK;

    _copyback_fmVector_Host( tau2, gCpuObj->tau2);
    
//printf("End of part2\n");
    return (0);

}


__global__ void b_part3(struct blsGPUobj* gCuda, int N, int P )
{    
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if(j < P)
    {
        double d_j = _V(gCuda->ad, j );
        double dMu_j = 0.0;
        int N_Aa = 0;

        for (int k=0;k<N;k++)
        {
             double pD_kj = _M(gCuda->gen_pD, k, j);
             double spdY_k = _V(gCuda->spdY, k );
             
             double x0 = _V( gCuda->tmp3, k) - spdY_k + pD_kj * d_j;
             dMu_j = dMu_j + x0 * pD_kj;

             if (pD_kj == 0) N_Aa++;
        }     
        
        dMu_j = dMu_j/ _V( gCuda->tmp, j);
             
        double dVar_j =  _V(gCuda->Var, j);
        
        //repalce rnorm with hiprand_normal_double
        //double new_d = rnorm( dMu_j, sqrt(dVar_j) );

        hiprandState s ;
        // seed a random number generator
        hiprand_init ( (unsigned int) j*10, 0, 0, &s) ;
        double _rn = hiprand_normal_double (&s);   
        double new_d = _rn * sqrt(dVar_j) + dMu_j;
        
        if (N_Aa == 0) 
            new_d = 0.0;

        _V(gCuda->ad, j) = new_d;
   }
}

int _cuda_bpart3( struct blsGPUobj* gCuda, struct blsGPUobj* gCpuObj, int N, int P, CFmVector& spdY, CFmVector& dVar, CFmVector& tmp, CFmVector& tmp3, CFmVector& d)
{
    _copy_fmVector_Device( gCpuObj->spdY, spdY);
    _copy_fmVector_Device( gCpuObj->tmp, tmp );
    _copy_fmVector_Device( gCpuObj->tmp3, tmp3 );
    _copy_fmVector_Device( gCpuObj->Var, dVar);
    _copy_fmVector_Device( gCpuObj->ad, d );
    
    b_part3<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(gCuda, N, P);
    hipDeviceSynchronize();
    ERRCHECK;

    _copyback_fmVector_Host( d, gCpuObj->ad);
    
//printf("End of part3\n");
    return (0);

}

int _cuda_bpart4( struct blsGPUobj* gCuda, struct blsGPUobj* gCpuObj, int P, double lambda_st2, double sigma2, CFmVector& d, CFmVector& tau2_st )
{
    _copy_fmVector_Device( gCpuObj->ad, d );
    
    b_part2<<< (P + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( gCuda, P, lambda_st2, sigma2  );
    hipDeviceSynchronize();
    ERRCHECK;

    _copyback_fmVector_Host( tau2_st, gCpuObj->tau2);
    
//printf("End of part4\n");
    return (0);
}
